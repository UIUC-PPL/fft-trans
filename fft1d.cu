#include "hip/hip_runtime.h"
#include "fft1d.h"

#define BLOCK_DIM 16

__global__ void twiddleKernel(complex_t* out, int N, int numChares, int k,
    double sign) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  double a, c, s, re, im;
  int idx = j*N+i;

  if (i < N && j < N/numChares) {
    a = sign * (TWOPI*(j+k*N/numChares)*i)/(N*N);
    c = cos(a);
    s = sin(a);
    re = c*out[idx].x - s*out[idx].y;
    im = s*out[idx].x + c*out[idx].y;
    out[idx].x = re;
    out[idx].y = im;
  }
}

void invokeTwiddle(complex_t* out, int N, int numChares, int k, double sign,
    hipStream_t stream) {
  dim3 block_dim(BLOCK_DIM, BLOCK_DIM);
  dim3 grid_dim((N+block_dim.x-1) / block_dim.x,
      (N/numChares+block_dim.y-1) / block_dim.y);

  twiddleKernel<<<grid_dim, block_dim, 0, stream>>>(out, N, numChares, k, sign);
  hapiCheck(hipPeekAtLastError());
}
